#include <iostream>
#include <hip/hip_runtime.h>
#include <vector>
#include <atomic>

#define PCG_DEFAULT_INC_64 1442695040888963407ULL
#define Math_TAU 6.2831853071795864769252867666
#define CMP_EPSILON 0.00001

typedef struct { uint64_t state;  uint64_t inc; } pcg32_random_t;
typedef struct {
    int character;
    int abilityCharacter;
    double abilityLevel;
    int itemCounts[8];
    double startTime;
    int32_t colorState;
    double intensity;
    // would do rgb but cba to figure out imports/packages/whatever or to make my own colour converter
} loadout;

// __device__
// bool seenSeeds[4294967296];

// std::vector<char> characterSet = {
//     '0', '1', '2', '2', '3', '4', '5', '6', '7', '8', '9',
//     'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z'
// };

// helper functions

__device__
double clamp(double m_a, double m_min, double m_max) {
	if (m_a < m_min) {
		return m_min;
	} else if (m_a > m_max) {
		return m_max;
	}
	return m_a;
}

__device__ float lerp(float a, float b, float t) {
    return a + t * (b - a);
}

__device__ void intToChar(uint32_t num, char* str, int maxLength) {
    int index = 0;
    while (num > 0 && index < maxLength - 1) {
        int digit = num % 10;
        str[index++] = '0' + digit;
        num /= 10;
    }
    if (index < maxLength) {
        str[index] = '\0';
    }
}

__device__ double pinch(double v) { // function run() uses
	if (v < 0.5) {
		return -v * v;
	}
	return v * v;
}

// could be: pinch(), 

__device__ double run(double x, double a, double b, double c) { // TorCurve.run() in windowkill
	c = pinch(c);
	x = fmaxf(0, fminf(1, x));
    const float eps = 0.00001f;
	double s = exp(a);
	double s2 = 1.0 / (s + eps);
	double t = fmaxf(0, fminf(1, b));
	double u = c;

	double res, c1, c2, c3;

	if (x < t) {
		c1 = (t * x) / (x + s*(t-x) + eps);
		c2 = t - pow(1/(t+eps), s2-1)*pow(abs(x-t), s2);
		c3 = pow(1/(t+eps), s-1) * pow(x, s);
	} else {
		c1 = (1-t)*(x-1)/(1-x-s*(t-x)+eps) + 1;
		c2 = pow(1/((1-t)+eps), s2-1)*pow(abs(x-t), s2) + t;
		c3 = 1 - pow(1/((1-t)+eps), s-1)*pow(1-x, s);
	}

	if (u <= 0) {
		res = (-u)*c2 + (1+u)*c1;
	} else {
		res = (u)*c3 + (1-u)*c1;
	}

	return res;
}
__device__
double smoothCorner(double x, double m, double l, double s) { // TorCurve.smoothCorner in windowkill
	double s1 = pow(s/10.0, 2.0);
	return 0.5 * ((l*x + m*(1.0+s1)) - sqrt(pow(abs(l*x-m*(1.0-s1)), 2.0)+4.0*m*m*s1));
}
// end of helper functions

// random number generator (pcg32)

__device__
uint32_t pcg32_random_r(pcg32_random_t* rng);
__device__
void pcg32_srandom_r(pcg32_random_t* rng, uint64_t initstate, uint64_t initseq);
__device__
uint32_t pcg32_boundedrand_r(pcg32_random_t* rng, uint32_t bound);

class RandomPCG {
	pcg32_random_t pcg;
	uint64_t current_seed = 0; // The seed the current generator state started from.
	uint64_t current_inc = 0;

public:
	static const uint64_t DEFAULT_SEED = 12047754176567800795U;
	static const uint64_t DEFAULT_INC = PCG_DEFAULT_INC_64;

    __device__
	RandomPCG(uint64_t p_seed = DEFAULT_SEED, uint64_t p_inc = DEFAULT_INC);

    __device__
	void seed(uint64_t p_seed) {
		current_seed = p_seed;
		pcg32_srandom_r(&pcg, current_seed, current_inc);
	}
    __device__
	uint64_t get_seed() { return current_seed; }

    __device__
	void set_state(uint64_t p_state) { pcg.state = p_state; }
    __device__
	uint64_t get_state() const { return pcg.state; }

    __device__
	uint32_t rand() {
		return pcg32_random_r(&pcg);
	}
    __device__
	uint32_t randbound(uint32_t bounds) {
		return pcg32_boundedrand_r(&pcg, bounds);
	}

    __device__
	double randd() {
		uint32_t proto_exp_offset = rand();
		if (proto_exp_offset == 0) {
			return 0;
		}
		uint64_t significand = (((uint64_t)rand()) << 32) | rand() | 0x8000000000000001U;
		return ldexp((double)significand, -64 - __clzll(proto_exp_offset));
	}
    __device__
	double randf() {
		uint32_t proto_exp_offset = rand();
		if (proto_exp_offset == 0) {
			return 0;
		}
		return (double) (float) (ldexp((double)(rand() | 0x80000001), -32 - __clz(proto_exp_offset)));
	}

    __device__
    double randfn(double p_mean, double p_deviation) {
        double temp = randf();
        if (temp < 0.00001) {
        temp += 0.00001;
    }
        return p_mean + p_deviation * (cos(6.2831853071795864769252867666 * static_cast<double>(randf())) * sqrt(-2.0 * log(static_cast<double>(temp))));
    }

    __device__
	double randomDouble(double p_from, double p_to);
    __device__
	double randomFloat(float p_from, float p_to);
    __device__
	int randomInteger(int p_from, int p_to);
};

__device__
RandomPCG::RandomPCG(uint64_t p_seed, uint64_t p_inc) :
		pcg(),
		current_inc(p_inc) {
	seed(p_seed);
}

__device__
double RandomPCG::randomDouble(double p_from, double p_to) {
	return randd() * (p_to - p_from) + p_from;
}

__device__
double RandomPCG::randomFloat(float p_from, float p_to) {
	return (double) (randf()*(p_to - p_from) + p_from);
}

__device__
int RandomPCG::randomInteger(int p_from, int p_to) {
	if (p_from == p_to) {
		return p_from;
	}
	return randbound(abs(p_from - p_to) + 1) + min(p_from, p_to);
}

__device__
uint32_t pcg32_random_r(pcg32_random_t* rng)
{
    uint64_t oldstate = rng->state;
    
    rng->state = oldstate * 6364136223846793005ULL + (rng->inc|1);
    
    uint32_t xorshifted = ((oldstate >> 18u) ^ oldstate) >> 27u;
    uint32_t rot = oldstate >> 59u;
    return (xorshifted >> rot) | (xorshifted << ((-rot) & 31));
}

__device__
void pcg32_srandom_r(pcg32_random_t* rng, uint64_t initstate, uint64_t initseq)
{
    rng->state = 0U;
    rng->inc = (initseq << 1u) | 1u;
    pcg32_random_r(rng);
    rng->state += initstate;
    pcg32_random_r(rng);
}

__device__
uint32_t pcg32_boundedrand_r(pcg32_random_t *rng, uint32_t bound) {
	uint32_t threshold = -bound % bound;

	for (;;) {
		uint32_t r = pcg32_random_r(rng);
		if (r >= threshold)
			return r % bound;
	}
}

class RandomNumberGenerator {
protected:
	RandomPCG randbase;
public:
    __device__
	void set_seed(uint64_t p_seed) { randbase.seed(p_seed); }
    __device__
	uint64_t get_seed() { return randbase.get_seed(); }

    __device__
	void set_state(uint64_t p_state) { randbase.set_state(p_state); }
    __device__
	uint64_t get_state() const { return randbase.get_state(); }

    __device__
	uint32_t randbound(uint32_t bounds) {
		return randbase.randbound(bounds);
	}
    __device__
	uint32_t randi() { return randbase.rand(); }
    __device__
	double randf() { return randbase.randf(); }
    __device__
	double randf_range(float p_from, float p_to) {
        return randbase.randomFloat(p_from, p_to);
    }
    __device__
	double randfn(float p_mean = 0.0, float p_deviation = 1.0) { return randbase.randfn(p_mean, p_deviation); }
    __device__
	int randi_range(int p_from, int p_to) { return randbase.randomInteger(p_from, p_to); }
    __device__
    void shuffle(int *arr, int n) {
        if (n <= 1) return;

        for (int i = n - 1; i > 0; i--) {
            int j = randbase.randbound(i + 1);
            
            int temp = arr[i];
            arr[i] = arr[j];
            arr[j] = temp;
        }
    }
};

// end of random number generator

// seed function

__device__
loadout get_results(uint64_t seed) {
    RandomNumberGenerator rng;
    RandomNumberGenerator globalRng;

    double itemCosts[8];
    itemCosts[0] = 1.0; // speed
    itemCosts[1] = 2.8; // fireRate
    itemCosts[2] = 3.3; // multiShot
    itemCosts[3] = 1.25; // wallPunch
    itemCosts[4] = 2.0; // splashDamage
    itemCosts[5] = 2.4; // piercing
    itemCosts[6] = 1.5; // freezing
    itemCosts[7] = 2.15; // infection

    int itemCategories[8];
    itemCategories[0] = 0; // speed
    itemCategories[1] = 1; // fireRate
    itemCategories[2] = 2; // multiShot
    itemCategories[3] = 3; // wallPunch
    itemCategories[4] = 4; // splashDamage
    itemCategories[5] = 5; // piercing
    itemCategories[6] = 6; // freezing
    itemCategories[7] = 7; // infection

    int charList[6];
    charList[0] = 0; // basic
    charList[1] = 1; // mage
    charList[2] = 2; // laser
    charList[3] = 3; // melee
    charList[4] = 4; // pointer
    charList[5] = 5; // swarm

    int itemCounts[8];

    rng.set_seed(seed);
    double intensity = rng.randf_range(0.20f, 1.0f);

    int character = charList[rng.randi() % 6];
    int abilityChar = charList[rng.randi() % 6];
    double abilityLevel = 1.0 + round(run(rng.randf(), 1.5/(1.0+intensity),1.0,0.0)*6);

    double itemCount = 8.0;


    double points = 0.66 * itemCount * rng.randf_range(0.5, 1.5) * (1.0 + 4.0*pow(intensity, 1.5));

    double itemDistSteepness = rng.randf_range(-0.5, 2.0);
    
    double itemDistArea = 1.0 / (1.0 + pow(2.0, 0.98*itemDistSteepness));

    globalRng.set_seed(rng.get_seed());
    globalRng.shuffle(itemCategories, 8);
    
    if (rng.randf() < intensity) {
        int multishotIdx = -1;
        for (int i = 0; i < itemCount; ++i) {
            if (itemCategories[i] == 2) {
                multishotIdx = i;
                break;
            }
        }

        if (multishotIdx != -1) {
            // Remove the multishot element
            for (int i = multishotIdx; i < itemCount - 1; ++i) {
                itemCategories[i] = itemCategories[i + 1];
            }
        }

        // Insert multiShot at a new index
        int insertIdx = itemCount - 1 - rng.randi_range(0, 2);
        for (int i = itemCount; i > insertIdx; --i) {
            itemCategories[i] = itemCategories[i - 1];
        }
        itemCategories[insertIdx] = 2;
    }

    if (rng.randf() < intensity) {
        int fireRateIdx = -1;
        for (int i = 0; i < itemCount; ++i) {
            if (itemCategories[i] == 1) {
                fireRateIdx = i;
                break;
            }
        }

        if (fireRateIdx != -1) {
            // Remove the firerate element
            for (int i = fireRateIdx; i < itemCount - 1; ++i) {
                itemCategories[i] = itemCategories[i + 1];
            }
        }

        // Insert firerate at a new index
        int insertIdx = itemCount - 1 - rng.randi_range(0, 2);
        for (int i = itemCount; i > insertIdx; --i) {
            itemCategories[i] = itemCategories[i - 1];
        }
        itemCategories[insertIdx] = 1;
    }

    double catMax = 7.0;
    // int total = 0; // why does this exist?
    for (int i = 0; i < 8; i++) {
        int item = itemCategories[i];
        double catT = (double) i / catMax;
        double cost = itemCosts[item];
        cost = 1.0 + ((cost - 1.0) / 2.5);
        double baseAmount = 0.0;

        double special = 0.0;
        if (i == 7) {
            special += 4.0 * rng.randf_range(0.0, pow(intensity, 2.0));
        }
        double amount = fmax(0.0, 3.0 * run(catT, itemDistSteepness, 1.0, 0.0) + 3.0 * clamp(rng.randfn(0.0, 0.15), -0.5, 0.5));
        
        itemCounts[item] = (int) clamp(round(baseAmount+amount*((points/cost)/(1.0+5.0*itemDistArea))+special), 0.0, 26.0);
    }

    intensity = -0.05 + intensity*lerp(0.33, 1.2, smoothCorner(((double) itemCounts[2]*1.8+(double) itemCounts[1])/12.0, 1.0, 1.0, 4.0)); // TODO: smoothCorner()

    double finalT = rng.randfn((float) pow(intensity, 1.2), 0.05);
    double startTime = clamp(lerp(60.0*2.0, 60.0*20.0, finalT), 60.0*2.0, 60.0*25.0);

    rng.randf();
    rng.randf();
    int colorState = rng.randi_range(0, 2);
    return loadout{character, abilityChar, abilityLevel, {itemCounts[0], itemCounts[1], itemCounts[2], itemCounts[3], itemCounts[4], itemCounts[5], itemCounts[6], itemCounts[7]}, startTime, colorState};
}

__device__ const char characterSet[36] = {
    '0', '1', '2', '3', '4', '5', '6', '7', '8', '9',
    'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 
    'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 
    'w', 'x', 'y', 'z'
};

__device__ void generateCharacterSequence(int index, char* result) {
    const int base = 36;
    int pos = 0;
    char temp[64]; // Temporary storage for reversed characters

    // Convert index to base-36 representation
    while (index > 0) {
        temp[pos++] = characterSet[(index - 1) % base]; // Map to character set (1-based index)
        index = (index - 1) / base; // Move to the next digit
    }
    
    // Reverse the characters into the result array
    for (int i = 0; i < pos; ++i) {
        result[i] = temp[pos - i - 1];
    }
    for (int i = pos; i < 14; ++i) {
        result[i] = '0';
    }

    result[14] = '\0'; // Null-terminate the result
}

__device__ uint32_t djb2Hash(const char *str) {
    unsigned long hash = 5381;
    int c;
    while (c = *str++) {
        hash = ((hash << 5) + hash) + c; /* hash * 33 + c */
    }

    return hash;
}

__device__ bool shouldStop = false;

__device__
void giveResults(loadout loadout, int seedsProcessed) {

}

__global__
void bruteForce(float clockRateKHz) {
    uint64_t start = clock64();
    int totalThreads = blockDim.x * gridDim.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    char result[14]; // Local buffer for the result
    int i = 0;
    for (;;i++) {
        generateCharacterSequence(idx + totalThreads * i + 1, result);
        uint32_t hash = djb2Hash(result);
        loadout loadout = get_results(hash);
        if (loadout.itemCounts[2] > 25 && loadout.itemCounts[0] > 25 && loadout.itemCounts[1] > 25) {
            shouldStop = true;

            uint64_t end = clock64();
            uint64_t elapsed = end - start;
            double timeInSeconds = (double) (elapsed / (clockRateKHz * 1000.0f));
            printf("Seconds passed:                              %.15f\n", timeInSeconds);
            printf("Rough estimate for seeds checked:            %d\n", idx+totalThreads*i+1);
            double timePerSeed = timeInSeconds/(double) (idx+totalThreads*i+1);
            printf("Rough estimate for time per seed in seconds: %.15f\n", timePerSeed);
            printf("Seed:                                        %s\n", result);
            printf("Hash:                                        %ld\n", hash);
            printf("\n");
        }
        if (shouldStop) {
            break;
        }
    }
}

int main() {
    printf("running\n");
    int device;

    // Get the current device
    hipGetDevice(&device);

    // Get the clock rate (in kHz)
    int clockRateKHz;
    hipDeviceGetAttribute(&clockRateKHz, hipDeviceAttributeClockRate, device);

    // Convert to kHz for kernel use
    float clockRate = (float)clockRateKHz;

    bruteForce<<<1024,256>>>(clockRate);
    
    hipDeviceSynchronize();

    // printf("%d %f %d %d %f %f", winningLoadout.abilityCharacter, winningLoadout.abilityLevel, winningLoadout.character, winningLoadout.colorState, winningLoadout.intensity,winningLoadout.startTime);

    return 0;
}